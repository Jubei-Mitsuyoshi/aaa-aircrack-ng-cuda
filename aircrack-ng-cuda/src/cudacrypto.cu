#include "hip/hip_runtime.h"
/*
 * WPA HASH - Cuda Implementations
 *
 * Copyright (C) 2009 Julian Tyler (tylerj@crm114.net)
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA
 */


#include "crypto.h" 
#include <string.h>

/* Cuda Constants */
#define	THREADS_PER_BLOCK	128

/* Size Constants */
#define PMK_SIZE		128
#define BUFFER_SIZE		65

/* Linear Memory Access Macros */
#define pmk_off(x) (pmk + (x * PMK_SIZE * sizeof(unsigned char)))
#define buffer_off(x) (buffer + (x * BUFFER_SIZE * sizeof(unsigned char)))
#define h_buffer_off(x) (h_buffer + (x * BUFFER_SIZE * sizeof(unsigned char)))

/* SHA1 CircularShift */
#define SHA1CircularShift(bits,word) 	((((word) << (bits))) | ((word) >> (32-(bits))))

/* Constants defined in SHA-1 */
#define D_K0	0x5A827999
#define D_K1	0x6ED9EBA1
#define D_K2	0x8F1BBCDC
#define D_K3	0xCA62C1D6

/* Texture Reference */
typedef unsigned int uint;
texture<uint4, 1, hipReadModeElementType> t_ctx;

/* Cuda SHA1 */
__global__ void cuda_DO_SHA1( uchar *buffer, uchar *pmk, uint pmk_off) 
{
	/* Thread & Block Info */
	const uint p = blockIdx.x * blockDim.x + threadIdx.x;
	const uint tid = threadIdx.x;

	/* Looping Vars */
	uint i, t;

	uint temp;
	uint W[80];

	uint A, B, C, D, E;
	uint h0, h1, h2, h3, h4;

	/* pmk cache */
	__shared__ unsigned char pmk_cache[20 * THREADS_PER_BLOCK];

	/* Load Cache From Device Memory */
	#pragma unroll
	for(t = 0; t < 20; t++)
		pmk_cache[tid * 20 + t] = pmk_off(p)[pmk_off + t];	

	/* End Byte and Padding */
	W[5] = 0x80000000;
	W[6] = 0;
	W[7] = 0;
	W[8] = 0;
	W[9] = 0;
	W[10] = 0;
	W[11] = 0;
	W[12] = 0;
	W[13] = 0;

	/* Size */
	W[14] = 0;
	W[15] = 0x200 + 8*20; 

	/* Load inital values */
	h0 = buffer_off(p)[0] << 24;
	h0 |= buffer_off(p)[1] << 16;
	h0 |= buffer_off(p)[2] << 8;
	h0 |= buffer_off(p)[3];

	h1 = buffer_off(p)[4] << 24;
	h1 |= buffer_off(p)[5] << 16;
	h1 |= buffer_off(p)[6] << 8;
	h1 |= buffer_off(p)[7];

	h2 = buffer_off(p)[8] << 24;
	h2 |= buffer_off(p)[9] << 16;
	h2 |= buffer_off(p)[10] << 8;
	h2 |= buffer_off(p)[11];

	h3 = buffer_off(p)[12] << 24;
	h3 |= buffer_off(p)[13] << 16;
	h3 |= buffer_off(p)[14] << 8;
	h3 |= buffer_off(p)[15];

	h4 = buffer_off(p)[16] << 24;
	h4 |= buffer_off(p)[17] << 16;
	h4 |= buffer_off(p)[18] << 8;
	h4 |= buffer_off(p)[19];

	#pragma unroll 
	for(i = 0; i < 4095; i++) {
    
		/* Inner Context */
		W[0] = h0;
		W[1] = h1;
		W[2] = h2;
		W[3] = h3;
		W[4] = h4;

		h0 = tex1Dfetch(t_ctx, p*3).x;
		h1 = tex1Dfetch(t_ctx, p*3).y;
		h2 = tex1Dfetch(t_ctx, p*3).z;
		h3 = tex1Dfetch(t_ctx, p*3).w;
		h4 = tex1Dfetch(t_ctx, p*3+2).x;
    
		#include "cudacrypto_sha1_process.cuh"

		/* Outer Context */
		W[0] = h0;
		W[1] = h1;
		W[2] = h2;
		W[3] = h3;
		W[4] = h4;

		h0 = tex1Dfetch(t_ctx, p*3+1).x;
		h1 = tex1Dfetch(t_ctx, p*3+1).y;
		h2 = tex1Dfetch(t_ctx, p*3+1).z;
		h3 = tex1Dfetch(t_ctx, p*3+1).w;
		h4 = tex1Dfetch(t_ctx, p*3+2).y;

		#include "cudacrypto_sha1_process.cuh"
 
		/* Cache update */
		pmk_cache[tid * 20 + 0] ^= h0 >> 24 & 0xFF;
		pmk_cache[tid * 20 + 1] ^= h0 >> 16 & 0xFF;
		pmk_cache[tid * 20 + 2] ^= h0 >> 8 & 0xFF;
		pmk_cache[tid * 20 + 3] ^= h0 & 0xFF;

		pmk_cache[tid * 20 + 4] ^= h1 >> 24 & 0xFF;
		pmk_cache[tid * 20 + 5] ^= h1 >> 16 & 0xFF;
 		pmk_cache[tid * 20 + 6] ^= h1 >> 8 & 0xFF;
		pmk_cache[tid * 20 + 7] ^= h1 & 0xFF;

		pmk_cache[tid * 20 + 8] ^= h2 >> 24 & 0xFF;
		pmk_cache[tid * 20 + 9] ^= h2 >> 16 & 0xFF;
		pmk_cache[tid * 20 + 10] ^= h2 >> 8 & 0xFF;
		pmk_cache[tid * 20 + 11] ^= h2 & 0xFF;

		pmk_cache[tid * 20 + 12] ^= h3 >> 24 & 0xFF;
		pmk_cache[tid * 20 + 13] ^= h3 >> 16 & 0xFF;
		pmk_cache[tid * 20 + 14] ^= h3 >> 8 & 0xFF;
		pmk_cache[tid * 20 + 15] ^= h3 & 0xFF;

		pmk_cache[tid * 20 + 16] ^= h4 >> 24 & 0xFF;
		pmk_cache[tid * 20 + 17] ^= h4 >> 16 & 0xFF;
		pmk_cache[tid * 20 + 18] ^= h4 >> 8 & 0xFF;
		pmk_cache[tid * 20 + 19] ^= h4 & 0xFF;
	}

	/* Copy result from Cache to Device Memory */
	#pragma unroll
	for(t = 0; t < 20; t++)
		pmk_off(p)[pmk_off + t] = pmk_cache[tid * 20 + t];
}

extern "C" void cuda_calc_pmk( int count, char **key, char *essid_pre, unsigned char **pmk ) {
	uint i, p, slen;
	char essid[33+4];

	/* Host Memory */
	uchar *h_pmk;
	uchar *h_buffer;
	uint4 *h_ctx;

	/* Device Memory */
        uchar *d_buffer;
        uchar *d_pmk;
	uint4 *d_ctx;

	/* Texture Reference */
	const textureReference* t_ctx_ptr;

	/* Temp SHA Contexts: inner & outer */
	SHA_CTX tmp_ctx_ipad;
	SHA_CTX tmp_ctx_opad;

	/* Allocate Host Memory */
	h_pmk = (uchar *)malloc(sizeof(uchar) * PMK_SIZE * count);
	h_buffer = (uchar *)malloc(sizeof(uchar) * BUFFER_SIZE * count);
	h_ctx = (uint4 *)malloc(sizeof(uint4) * count * 3);	// 10 uints in 3 uint4

	/* Allocate Device Memory */
	hipMalloc((void **)&d_pmk, sizeof(uchar) * count * 128);
	hipMalloc((void **)&d_buffer, sizeof(char) * 65 * count);

	/* Setup Texture Reference */
	hipGetTextureReference(&t_ctx_ptr, HIP_SYMBOL("t_ctx"));
	hipMalloc((void **)&d_ctx, count * sizeof(uint4) * 3);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uint4>(); 
	hipBindTexture(0, t_ctx_ptr, d_ctx, &channelDesc, count * sizeof(uint4) * 3);

	/* Setup Block and Grid */
	dim3 dimBlock(THREADS_PER_BLOCK,1);
	dim3 dimGrid((count+dimBlock.x-1)/dimBlock.x, 1);
	
	memset(essid, 0, sizeof(essid));
	memcpy(essid, essid_pre, strlen(essid_pre)+1);
	slen = strlen( essid ) + 4;

	/* setup the inner and outer contexts */
	for(p = 0; p < count; p++) {

		/* Inner Context */
		memset( h_buffer_off(p), 0, sizeof(uchar) * BUFFER_SIZE);
		strncpy( (char *) (h_buffer_off(p)), key[p], BUFFER_SIZE - 1 );

		for( i = 0; i < 64; i++ )
			h_buffer_off(p)[i] ^= 0x36;

		SHA1_Init( &(tmp_ctx_ipad) );
		SHA1_Update( &(tmp_ctx_ipad), h_buffer_off(p), 64 );
		
		h_ctx[3*p].x = tmp_ctx_ipad.h0;
		h_ctx[3*p].y = tmp_ctx_ipad.h1;
		h_ctx[3*p].z = tmp_ctx_ipad.h2;
		h_ctx[3*p].w = tmp_ctx_ipad.h3;
		h_ctx[3*p+2].x = tmp_ctx_ipad.h4;


		/* Outer Context */
		for( i = 0; i < 64; i++ )
			h_buffer_off(p)[i] ^= 0x6A;

		SHA1_Init( &(tmp_ctx_opad) );
		SHA1_Update( &(tmp_ctx_opad), h_buffer_off(p), 64 );

		h_ctx[3*p+1].x = tmp_ctx_opad.h0;
		h_ctx[3*p+1].y = tmp_ctx_opad.h1;
		h_ctx[3*p+1].z = tmp_ctx_opad.h2;
		h_ctx[3*p+1].w = tmp_ctx_opad.h3;
		h_ctx[3*p+2].y = tmp_ctx_opad.h4;

		/* iterate HMAC-SHA1 over itself 8192 times */
		essid[slen - 1] = '\1';
		HMAC(EVP_sha1(), (uchar *)key[p], strlen(key[p]), (uchar*)essid, slen, pmk[p], NULL);
		memcpy( h_buffer_off(p), pmk[p], 20 );
	}
		

	/* Copy from (array of pointers to string) to (linear array of strings) */
	for(i = 0; i < count; i++) {
		uchar *dest =  h_pmk + (i * PMK_SIZE);
		memcpy(dest, pmk[i], PMK_SIZE);
	}

	/* Copy to Device Memory */
	hipMemcpy(d_ctx, h_ctx, count * sizeof(uint4) * 3, hipMemcpyHostToDevice);
	hipMemcpy(d_pmk, h_pmk, count * 128 * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy( d_buffer, h_buffer, count * 65 * sizeof(char), hipMemcpyHostToDevice);

	/* Do First 4096 iterations of SHA1 */
	cuda_DO_SHA1<<<dimGrid, dimBlock>>>(d_buffer, d_pmk, 0);
	
	/* Copy to result */
	hipMemcpy(h_pmk, d_pmk, count * 128 * sizeof(char), hipMemcpyDeviceToHost);

	/* Digest on linear memory */
	for(p = 0; p < count; p++) {
		essid[slen - 1] = '\2';
		HMAC(EVP_sha1(), (uchar *)(key[p]), strlen(key[p]), (uchar*)essid, slen, (h_pmk + (PMK_SIZE * p)) + 20, NULL);
		memcpy( h_buffer_off(p), (h_pmk + (PMK_SIZE * p)) + 20, 20 );
	}

	/* Copy back to device memory */
	hipMemcpy(d_pmk, h_pmk, count * 128 * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy( d_buffer, h_buffer, count * 65 * sizeof(char), hipMemcpyHostToDevice);

	/* Do Second 4096 iterations of SHA1 */
	cuda_DO_SHA1<<<dimGrid, dimBlock>>>(d_buffer, d_pmk, 20);

	/* Copy result from device memory */
	hipMemcpy(h_pmk, d_pmk, count * 128 * sizeof(char), hipMemcpyDeviceToHost);

	/* Convert result to (array of pointers to string) */
	for(i = 0; i < count; i++) {
		uchar *src = h_pmk + (i * PMK_SIZE);
		memcpy(pmk[i], src, PMK_SIZE);
	}


	/* Free Device Memory */
	hipFree(d_ctx);
	hipFree(d_pmk);
	hipFree(d_buffer);

	/* Free Host Memory */
	free(h_pmk);
	free(h_buffer);
	free(h_ctx);
}

extern "C" int cuda_getblocksize()
{
	int device_count;
	int blocksize = 0;
	hipDeviceProp_t device_prop;
	
	hipGetDeviceCount(&device_count);

	/* No Cuda Device Found */
	if(device_count == 0)
		return 0;

        hipGetDeviceProperties(&device_prop, 0);

	/* Limited By Shared Memory */
	/* 5 Blocks per MP */
	/* (2588+28) * 5 ==  13080 < (16384 shared memory of per MP)*/

	blocksize = THREADS_PER_BLOCK * 5 * device_prop.multiProcessorCount;

	return blocksize;
}
